template<typename T, typename T_ACC>
void LaunchGammaBetaBackwardCUDAKernel(
    const T* dY_data, const T* X_data, const T_ACC* mean_data, const T_ACC* rstd_data, int64_t M, int64_t N, Tensor* dgamma, Tensor* dbeta, hipStream_t hip_stream) {
   ...
  if (M > 64 * 1024 && N / block_dim_x < sm_count / 2) {
  //Compute partial sums for the gradient in the first pass
  //Then do a .sum(0) to do a final reduction.
   ...
} else {
// Normal main Loop
}
