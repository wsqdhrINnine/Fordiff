template<typename T, typename T_ACC>
void LaunchGammaBetaBackwardCUDAKernel(
    const T* dY_data, const T* X_data, const T_ACC* mean_data, const T_ACC* rstd_data, int64_t M, int64_t N, Tensor* dgamma, Tensor* dbeta, hipStream_t hip_stream) {
  constexpr int block_dim_x = 32;
  const int sm_count = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  if (M > 64 * 1024 && N / block_dim_x < sm_count / 2) {
  //Compute partial sums for the gradient in the first pass
  //Then do a .sum(0) to do a final reduction.
   ...
} else {
// Normal main Loop
}
