#include "hip/hip_runtime.h"
template <typename T, typename T_ACC>
__global__ void GammaBetaBackwardCUDAKernel(
   int64_t M, int64_t N, const T* dY, const T* X, const T_ACC* mean, const T_ACC* rstd, T* dg, T* db) {
  alignas(sizeof(double)) extern __shared__ char s_data1[];
  T_ACC* s_data_typed = reinterpret_cast<T_ACC*>(&s_data1);
  T_ACC* s_dg;
  T_ACC* s_db;
  const int64_t j = ((int64_t) blockIdx.x) * blockDim.x + threadIdx.x;
  T_ACC dg_sum = 0;
  T_ACC db_sum = 0;
  if (j < N) {
    constexpr int unroll_factor = 8;
    T_ACC mean_reg;
    T_ACC rstd_reg;
    T dY_reg;
    T X_reg;
    // Main Loop
      ...
  }
